#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BILLION 1000000000.0
int array_size;
int test_count;
int block_size = 16;

void init_matrix(float *, int);
__global__ void matrix_add(float *, float *, float *);
bool matricesSumChecker(float *, float *, float *, int);

int main(int argc, char **argv)
{

    if (argc != 4)
    {
        printf("Kullanim: %s <test_count> <block_size> <array_size>\n", argv[0]);
        exit(EXIT_FAILURE);
    }
    // Parse the arguments
    test_count = atoi(argv[1]);
    array_size = atoi(argv[3]);

    // Define matrix size
    int size = array_size * array_size * array_size;

    // Allocate memory for matrices on the device
    float *d_A, *d_B, *d_C1, *d_C2;
    hipMalloc((void **)&d_A, size * sizeof(float));
    hipMalloc((void **)&d_B, size * sizeof(float));
    hipMalloc((void **)&d_C1, size * sizeof(float));
    hipMalloc((void **)&d_C2, size * sizeof(float));

    // Initialize matrices with random values
    float *h_A = (float *)malloc(size * sizeof(float));
    float *h_B = (float *)malloc(size * sizeof(float));

    init_matrix(h_A, size);
    init_matrix(h_B, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

    // Calculate number of blocks needed for matrix addition
    int num_blocks = (size + block_size - 1) / block_size;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);
    for (int i = 0; i < test_count; i++)
    {
        // Call kernel function to perform matrix addition in parallel
        matrix_add<<<num_blocks, block_size>>>(d_A, d_B, d_C1);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    ms /= test_count;
    printf("Paralel Execution time: %f ms\n", ms);

    // Copy result matrix from device to host
    float *h_C1 = (float *)malloc(size * sizeof(float));
    hipMemcpy(h_C1, d_C1, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Check parallel matrices sum: ");
    if (matricesSumChecker(h_A, h_B, h_C1, array_size))
        printf("SUCCESS\n");
    else
        printf("FAILED\n");

    // Serial
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);
    for (int i = 0; i < test_count; i++)
    {
        // Call kernel function to perform matrix addition in serial
        matrix_add<<<1, size>>>(d_A, d_B, d_C2);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    ms /= test_count;
    printf("Serial Execution time: %f ms\n", ms);

    // Copy result matrix from device to host
    float *h_C2 = (float *)malloc(size * sizeof(float));
    hipMemcpy(h_C2, d_C2, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Check matrices: ");
    if (matricesSumChecker(h_A, h_B, h_C2, array_size))
        printf("SUCCESS\n");
    else
        printf("FAILED\n");

    // Free memory on host and device
    free(h_A);
    free(h_B);
    free(h_C1);
    free(h_C2);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C1);
    hipFree(d_C2);

    return 0;
}

// Function to initialize matrices with random values
void init_matrix(float *matrix, int size)
{
    for (int i = 0; i < size; i++)
    {
        matrix[i] = (float)rand() / RAND_MAX;
    }
}

// Kernel function to perform  matrix addition
__global__ void matrix_add(float *A, float *B, float *C)
{
    // Calculate global index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Add corresponding elements of A and B and store result in C
    C[idx] = A[idx] + B[idx];
}

bool matricesSumChecker(float *M1, float *M2, float *MResult, int array_size)
{
    float *MSum = (float *)malloc(array_size * array_size * array_size * sizeof(float));
    for (int i = 0; i < array_size; i++)
    {
        for (int j = 0; j < array_size; j++)
        {
            for (int k = 0; k < array_size; k++)
            {
                MSum[i * array_size * array_size + j * array_size + k] = M1[i * array_size * array_size + j * array_size + k] + M2[i * array_size * array_size + j * array_size + k];
                if (fabs(MSum[i * array_size * array_size + j * array_size + k] - MResult[i * array_size * array_size + j * array_size + k]) > 1e-5)
                {
                    return false;
                }
            }
        }
    }
    return true;
}